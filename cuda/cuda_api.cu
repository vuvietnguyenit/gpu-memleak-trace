// cuda_api.cu
#include <hip/hip_runtime.h>
#include <string.h>

extern "C" int getDeviceName(int deviceID, char *nameBuffer,
                             size_t bufferSize) {
  struct hipDeviceProp_t prop; // Use struct for C compatibility
  hipError_t err = hipGetDeviceProperties(&prop, deviceID);
  if (err != hipSuccess) {
    return (int)err;
  }
  strncpy(nameBuffer, prop.name, bufferSize - 1);
  nameBuffer[bufferSize - 1] = '\0';
  return 0; // success
}

extern "C" const char *checkCuda(hipError_t err) {
  if (err != hipSuccess) {
    return hipGetErrorString(err);
  }
  return NULL;
}
